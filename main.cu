#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"


__global__ void create_world(hitable **d_esferas, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_esferas)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_esferas+1) = new sphere(vec3(0,-100.5,-1), 100);
        *d_world    = new hitable_list(d_esferas,2);
    }
}

__device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f*(unit_direction.y() + 1.0f);
        return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}
__global__ void render(vec3 *fb, int max_x, int max_y,
                       vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin,
                       hitable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r, world);
}
int main(){

  int nx = 1200;
  int ny = 600;

  //arquivo de saida
  std::ofstream myfile;
  myfile.open ("image.ppm");
  myfile << "P3\n" << nx << " " << ny << "\n255\n";

  hipError_t error;

  int resolution = nx*ny;
  int color_channel = 3;
  int img_buffer_size = resolution*color_channel*sizeof(float);

  // allocate img_buffer
  vec3 *img_buffer;
  error = hipMallocManaged((void **)&img_buffer, img_buffer_size);
  if(error!=hipSuccess) {
        printf("Memory Allocation CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));
        exit(EXIT_FAILURE);
  }


  //alocando espaço para as esferas
  hitable **d_esferas;
  error = hipMalloc((void **)&d_esferas, 2*sizeof(hitable *));
  if(error!=hipSuccess) {
      printf("Memory Allocation CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }

  //alocando espaço para o mundo
  hitable **d_world;
  error = hipMalloc((void **)&d_world, sizeof(hitable *));
  if(error!=hipSuccess) {
      printf("Memory Allocation CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));
      exit(EXIT_FAILURE);
  }

  //criando o mundo 
  create_world<<<1,1>>>(d_esferas,d_world);

  hipDeviceSynchronize();



  float block_size = 16.0;
  // Dimensoes para organizar na GPU
  dim3 dimGrid(ceil(ny/block_size), ceil(nx/block_size), 1);
  dim3 dimBlock((int) block_size, int (block_size), 1);


  render<<<dimGrid, dimBlock>>>(img_buffer, nx, ny,
                            vec3(-2.0, -1.0, -1.0),
                            vec3(4.0, 0.0, 0.0),
                            vec3(0.0, 2.0, 0.0),
                            vec3(0.0, 0.0, 0.0),
                            d_world);

  hipDeviceSynchronize();

  //jogando os pixels calculado para o arquivo de saida formador da imagem
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
        size_t pixel_index = j*nx + i;
        int ir = int(255.99*img_buffer[pixel_index].r());
        int ig = int(255.99*img_buffer[pixel_index].g());
        int ib = int(255.99*img_buffer[pixel_index].b());
        myfile << ir << " " << ig << " " << ib << "\n";
    }
  }
  
  hipDeviceReset();
}

